#include "hip/hip_runtime.h"
/* This file contains an example showing matrix multiplication using cuBLAS, 
   i.e., C(m, n) = alpha * A(m, k) %*% B(k, n) + beta * C(m, n); instead of passing 
   existing values, for A and B, we use cuRAND to generate random numbers to fill 
   the array */

/* The example is adapted from 
   https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/ */

/* necessary header files */
#include <R.h>
#include <ctime>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h> 
#include <hipblas.h>

/* fill the array A (nr_rows_A, nr_cols_A) with random numbers on GPU */
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A)
{
  /* create a pseudo-random number generator */
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  
  /* set the seed for the random number generator using the system clock */
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
  
  /* fill the array with random numbers on the device */
  hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

/* Multiply the arrays A and B on GPU and save the result in C */
/* C(m, n) = alpha * A(m, k) %*% B(k, n) + beta * C(m, n) */
void gpu_blas_mmul(float *A, float *B, float *C, int m, int k, int n)
{
  int lda = m, ldb = k, ldc = m;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;
  
  /* create a handle for cuBLAS */
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  
  /* do the actual multiplication */
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
  
  /* destroy the handle */
  hipblasDestroy(handle);
}

extern "C"
/* cublas: this function performs matrix multiplication on GPU and send the result back to CPU */
/* C(m, n) = alpha * A(m, k) %*% B(k, n) + beta * C(m, n) */
void cublas(int *m, int *k, int *n, double *C)
{

  /* allocate 3 arrays on GPU(device) */
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, *m * *k * sizeof(float));
  hipMalloc(&d_B, *k * *n * sizeof(float));
  hipMalloc(&d_C, *m * *n * sizeof(float));
  
  /* Copy CPU data (i.e., 'C' in this example) to GPU */
  hipMemcpy(d_C, C, *m * *n * sizeof(float), hipMemcpyHostToDevice);
  
  /* fill the arrays A and B on GPU with random numbers */
  GPU_fill_rand(d_A, *m, *k);
  GPU_fill_rand(d_B, *k, *n);
  
  /* matrix multiplication on GPU */
  gpu_blas_mmul(d_A, d_B, d_C, *m, *k, *n);
  
  /* copy the result from device to host memory */
  hipMemcpy(C, d_C, *m * *n * sizeof(float), hipMemcpyDeviceToHost);
  
  /* free GPU memory */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  
}
